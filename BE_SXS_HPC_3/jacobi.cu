// nvcc jcaobi.cu -o jacobi


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define NBLOCKS 32

__global__ void compute_xnext(double *r, double *d, double *x, double *x_next,
	double *b, int size) {

	// Size defined as third arg in <<< >>> thing
	extern __shared__ double x_copy[];

	// Copy x to multiply in shared memory for quicker access
	int idx, row_idx = blockDim.x * blockIdx.x + threadIdx.x;
  int upBound = NBLOCKS * (threadIdx.x + 1);
  if (upBound > size)
    upBound = size;
  for (idx=NBLOCKS*threadIdx.x; idx < upBound; idx++)
    x_copy[idx] = x[idx];

	if (row_idx >= size) return;

	__syncthreads();

	double y_k = 0.;
	for (idx=0; idx < size; idx++)
		y_k += x_copy[idx] * r[row_idx * size + idx];

	x_next[row_idx] = (b[row_idx] - y_k) / d[row_idx];
}

int main (int argc, char *argvs[]) {

	if (argc != 4) {
		printf("usage: %s [size] [epsilon] [verbose: 1/2]\n", argvs[0]);
		return 1;
	}

	// init constants & variables
	int nBlocks, nThreadsPerBlock, size, idx1, idx2, niter, epsilon_pow, verbose;
	size = atoi(argvs[1]);
	epsilon_pow = atoi(argvs[2]);
	verbose = atoi(argvs[3]);
	nBlocks = NBLOCKS;
	nThreadsPerBlock = (int) size / nBlocks + 1;
	niter = 0;
	double epsilon = 1.;
	for (idx1=0; idx1 < epsilon_pow; idx1++) epsilon = epsilon / 10.;
	double epsilon2 = epsilon * epsilon;
	double norm2 = epsilon2 + 1.; // to be init over espilon

	struct timeval t0, t1, t2, t3;
	size_t size_mat = size * size * sizeof(double);
	size_t size_vect = size * sizeof(double);

	// init matrices
	double *r_mat, *d_vect, *x, *x_next, *b;
	double *d_r, *d_d, *d_x, *d_xnext, *d_b;
	r_mat = (double *) calloc(size * size, sizeof(double));
	d_vect = (double *) calloc(size, sizeof(double));
	x = (double *) calloc(size, sizeof(double));
	x_next = (double *) calloc(size, sizeof(double));
	b = (double *) calloc(size, sizeof(double));

	for (idx1=0; idx1<size; idx1++) {
		for (idx2=0; idx2<size; idx2++)
			r_mat[idx1*size + idx2] = 1.;
		r_mat[idx1*size + idx1] = 0.;
		d_vect[idx1] = 2. * size + 1.;
		x[idx1] = 1.;
		b[idx1] = 3.;
	}

	// Allocation & copy on device
	gettimeofday(&t0,NULL);
	hipMalloc(&d_r, size_mat);
	hipMalloc(&d_d, size_vect);
	hipMalloc(&d_x, size_vect);
	hipMalloc(&d_xnext, size_vect);
	hipMalloc(&d_b, size_vect);

	gettimeofday(&t1,NULL);
	hipMemcpy(d_r, r_mat, size_mat, hipMemcpyHostToDevice);
	hipMemcpy(d_d, d_vect, size_vect, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size_vect, hipMemcpyHostToDevice);

	gettimeofday(&t2,NULL);

	// Main Loop
	while (norm2 > epsilon2) {
		niter ++;
		// Send x_k
		hipMemcpy(d_x, x, size_vect, hipMemcpyHostToDevice);
		compute_xnext <<<nBlocks, nThreadsPerBlock, size_vect >>> (d_r, d_d,
			d_x, d_xnext, d_b, size);
			// Gather x_k+1
		hipMemcpy(x_next, d_xnext, size_vect, hipMemcpyDeviceToHost);

		norm2 = 0.;
		for (idx1=0; idx1 < size; idx1++) {
			norm2 += (x[idx1] - x_next[idx1]) * (x[idx1] - x_next[idx1]);
			x[idx1] = x_next[idx1];
		}
	}

	gettimeofday(&t3,NULL);

	hipFree(d_r);
	hipFree(d_d);
	hipFree(d_x);
	hipFree(d_xnext);
	hipFree(d_b);

	// Verbose
	double t_alloc = (double)(t1.tv_sec-t0.tv_sec) + \
		(double)(t1.tv_usec-t0.tv_usec)/1000000;
	double t_trans = (double)(t2.tv_sec-t1.tv_sec) + \
		(double)(t2.tv_usec-t1.tv_usec)/1000000;
	double t_calc  = (double)(t3.tv_sec-t2.tv_sec) + \
		(double)(t3.tv_usec-t2.tv_usec)/1000000;

	if (verbose == 1) {
		printf("N = %d\nEpsilon = %10.9f\n", size, epsilon);
		printf("Nombre d'iterations: %d\n", niter);
		double norm_error = 0.;
		for (int i=0; i < size; i++)
			norm_error += (x[i] - 1./size) * (x[i] - 1./size);
		printf("Error compared to sol.: %25.24f\n\n", norm_error);
		printf("Temps d'alloc. device   : %f s\n", t_alloc);
		printf("Temps de transfert init.: %f s\n", t_trans);
		printf("Temps de calcul:          %f s\n", t_calc);
	} else if (verbose == 2)
		printf("%d, %12.11f, %d, %f\n", size, epsilon, niter, t_calc);

	free(r_mat);
	free(d_vect);
	free(x);
	free(x_next);
	free(b);

	return 0;
}
